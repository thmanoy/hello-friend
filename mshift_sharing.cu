#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>
#define N 600
#define DIMS 2
#define epsilon 0.1
#define ss 1
#define blocksize 16
#define BLK (blocksize*blocksize)
#define dimGrid (N / BLK + 1)

__device__ float distance(float *a, float *b) {
    int i;
    float d = 0.0, t;
    for(i=0;i<DIMS;i++) {
        t = a[i] - b[i];
        d += t*t;
    }
    return d;
}

__device__ void meanshift(float *x, float *y, float *result) {
    int j, i;
    float sum1[DIMS], sum2 = 0.0, dis, k;
    for (i=0;i<DIMS;i++) sum1[i] = 0.0;
    for(j=0;j<N;j++) {
        dis = distance(y,x+DIMS*j);
        if (dis > ss*ss) continue;
        k = exp( - dis / (2*ss) );
        sum2 += k;
        for(i=0;i<DIMS;i++) sum1[i] += k * x[DIMS*j+i];
    }
    for(i=0;i<DIMS;i++) result[i] = sum1[i] / sum2;
}

__global__ void msfunction(float *x, float *y) {
    __shared__ float ys[BLK*DIMS], temp[BLK*DIMS], m[BLK];
    int h, j = threadIdx.x * blockDim.x  + threadIdx.y;
    int i = blockIdx.x * BLK + j;
    if (i < N) {
        memcpy(ys+DIMS*j, x+DIMS*i, DIMS*sizeof(float));
        do {
            meanshift(x, ys+DIMS*j, temp+DIMS*j);
            m[j] = sqrt(distance(temp+DIMS*j, ys+DIMS*j));
            memcpy(ys+DIMS*j, temp+DIMS*j, DIMS*sizeof(float));
        } while (m[j]>epsilon);
        for(h=0;h<DIMS;h++)
            y[DIMS*i+h] = ys[DIMS*j+h];
    }
}

int main() {
    dim3 dimBlock( blocksize, blocksize );
    int i, j, errors = 0;
    float *x, *y, *xd, *yd, msecs, temp;
    clock_t start, end;
    x = (float *)malloc(DIMS*N*sizeof(float));
    y = (float *)malloc(DIMS*N*sizeof(float));
    FILE *data = fopen("data.txt", "r"), *output = fopen("meanshift.txt", "w"), *check = fopen("results.txt", "r");
    for (i=0;i<DIMS*N;i++)
        fscanf(data, "%f", &x[i]);
    fclose(data);
    hipMalloc(&xd, N*DIMS*sizeof(float));
    hipMalloc(&yd, N*DIMS*sizeof(float));
    hipMemcpy(xd, x, N*DIMS*sizeof(float),hipMemcpyHostToDevice);
    start = clock();
    msfunction<<<dimGrid,dimBlock>>>(xd, yd);
    hipDeviceSynchronize();
    end = clock();
    msecs = (float)1000*(end - start) / CLOCKS_PER_SEC;
    hipMemcpy(y, yd, N*DIMS*sizeof(float),hipMemcpyDeviceToHost);
    for (i=0;i<N;i++)
        for (j=0;j<DIMS;j++) {
            if (j==DIMS-1) fprintf(output, "%f\n", y[DIMS*i+j]);
            else fprintf(output, "%f ", y[DIMS*i+j]);
        }
    for (i=0;i<N*DIMS;i++) {
        fscanf(check, "%f", &temp);
        if ( fabs( temp - y[i] ) > 0.1) errors++;
    }
    printf("Time is %f msecs\n", msecs);
    printf("detected %d errors, %.2f %% of the total values\n", errors, (float)100*errors/(N*DIMS));
    fclose(output);
    fclose(check);
    hipFree(xd);
    hipFree(yd);
    free(x);
    free(y);
}